#include "hip/hip_runtime.h"
#include <CImg.h>
#include <math.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>

using namespace cimg_library;

void sum_cpu(float* sum, CImg< unsigned char >* inputImage, int height, int width)
{
    for ( int y = 0; y < height; y++ ) {
      for ( int x = 0; x < width; x++ ) {
        float r = static_cast< float >(inputImage[(y * width) + x]);

        sum += r;
        }
     }
}

 __global__ void sum_gpu(float* sum, CImg< unsigned char >* inputImage, int height, int width)
{
    unsigned int x = (blockIdx.x * blockDim.x) + threadIdx.x;
    unsigned int y = (blockIdx.y * blockDim.y) + threadIdx.y;
    if ((x < width) && (y < height)){ 
      float r = static_cast< float >(inputImage[(y * width) + x]);

      sum += r;
      }
}

int main(int argc, char *argv[]){
  hipEvent_t start, stop;
  hipEventCreate(&start);
  hipEventCreate(&stop);
  float millisecondsCpu = 0;
  float millisecondsGpu = 0;
  
  CImg<unsigned char> img(640,400,1,3);

  float *img, *imgf, *kernel;
  
  img = (float*)malloc(Nx*Ny*sizeof(float));
  imgf = (float*)malloc(Nx*Ny*sizeof(float));
  kernel = (float*)malloc(kernel_size*kernel_size*sizeof(float));  
  
  
  float *d_img, *d_imgf, *d_kernel;
  
  hipMalloc(&d_img,Nx*Ny*sizeof(float));
  hipMalloc(&d_imgf,Nx*Ny*sizeof(float));
  hipMalloc(&d_kernel,kernel_size*kernel_size*sizeof(float));
  
  load_image(finput, Nx, Ny, img);
  calculate_kernel(kernel_size, sigma, kernel);

  hipMemcpy(d_img, img, Nx*Ny*sizeof(float),hipMemcpyHostToDevice);
  hipMemcpy(d_kernel,kernel, kernel_size*kernel_size*sizeof(float),hipMemcpyHostToDevice);

  Nblocks = Ny - (kernel_size-1);
  Nthreads = Nx - (kernel_size-1);
  
  hipEventRecord(start);
  sum_cpu(img, kernel, imgf, Nx, Ny, kernel_size);
  hipEventRecord(stop);
  hipEventElapsedTime(&millisecondsCpu, start, stop);

  printf("\n");
  printf("Ellapsed Time (CPU): %16.10f ms\n", millisecondsCpu);
  printf("\n");

  hipEventRecord(start);
  conv_img_gpu<<<Nblocks, Nthreads, kernel_size*kernel_size*sizeof(float)>>>(d_img, d_kernel, d_imgf, Nx, Ny, kernel_size);
  hipDeviceSynchronize();
  hipEventRecord(stop);
  hipEventElapsedTime(&millisecondsGpu, start, stop);
  
  hipMemcpy(imgf, d_imgf, Nx*Ny*sizeof(float), hipMemcpyDeviceToHost);
  save_image(foutput, Nx, Ny, imgf);
  
  printf("\n");
  printf("Ellapsed Time (GPU): %16.10f ms\n", millisecondsGpu);
  printf("\n");
  
  
  free(img);
  free(imgf);
  free(kernel);

  hipFree(d_img);
  hipFree(d_imgf);
  hipFree(d_kernel);
}